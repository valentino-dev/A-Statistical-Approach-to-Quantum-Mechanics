
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

// setup settings
__device__ __constant__ size_t d_n = 10; // Hits per Site

// min 2 and has to be the same
__device__ __constant__ size_t d_xjPerThread = 1 << 1;
size_t h_xjPerThread = 1 << 1;

__device__ __constant__ double d_a = 1.0;
__device__ __constant__ double d_lambda = 0.0;
__device__ __constant__ double d_mu_sq = 1.0;
__device__ __constant__ double d_f_sq = 2.0;
__device__ __constant__ double d_m0 = 1.0;
__device__ long seed = 0;
size_t runs = 4; // Runs (increase to yield more data)
size_t SII = 5;  // Statistical Independent Iterations
size_t RS = 0;   // Recording Start
char DATA_PATH[] = "data/data_fig4.csv";

// Guide for Device Setting:
// RTX 2060 super has 34 SMs
// 1 Warp are 32 Threads, 64 Warps per SM at maximum
// So 32*64*34=69,632 Threads can be active at once
// 512 or 1024 Threads per Block are ideal, though at 1024 it is harder to start

// Device dependent settings: do not change
//size_t MCS = 34 * 4 * 1 << 0; 		// 34SMs*4BlocksPerSM=136Blocks: Monte Carlo Simulations => 69,632 Threads => 1 Full GPU
size_t MCS = 1;
size_t MCI = 500;			// 500MCI/5SII=100 Configurations
size_t threadsPerBlock = 1 << 2; 	// 512 threads per block => 4 Blocks = 2048 Threads = 1 Full SM

size_t blocksPerGrid = MCS;

// to increase N, use h_xjPerThread
size_t N = h_xjPerThread * (threadsPerBlock) + 2;

__device__ size_t d_N = (1 << 3) * (1 << 9) + 2;

size_t length = MCS * N;
size_t size = length * sizeof(double);
size_t total_configurations = (MCI - RS) * MCS / SII;

#define gpuErrchk(ans)                                                         \
{ gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
		bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}
__global__ void init_rand(hiprandState *state) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(42, idx, 0, &state[idx]);
}

__device__ double rand_x(hiprandState *state, double min, double max) {
	return hiprand_uniform(state) * (max - min) + min;
}

__device__ double potential_1(double *x) {
	return (d_mu_sq * *x * *x * 0.5 + d_lambda * *x * *x * *x * *x);
}

__device__ double potential_2(double *x) {
	return d_lambda * pow(*x * *x - d_f_sq, 2);
}

__device__ double calc_S_of_xj(double x, double *ptr) {
	return d_m0 * 0.5 * (pow(x - *(ptr + 1), 2) + pow(*(ptr - 1) - x, 2)) / d_a +
		d_a * (potential_1(&x) + potential_1(ptr - 1));
}

__device__ double calc_tot_S(double *sites) {
	double temp = 0;
	for (size_t i = 0; i < d_N / 2 - 1; i++) {
		temp += calc_S_of_xj(sites[i * 2 + 1], sites + i * 2);
	}
	return temp;
}

__device__ double calc_dS(double *xptr, double *new_xptr) {
	return calc_S_of_xj(*new_xptr, xptr) - calc_S_of_xj(*xptr, xptr);
}

__device__ void step(double *xptr, hiprandState *local_state) {
	double delta = 2 * sqrt(d_a);
	double new_xptr = rand_x(local_state, *xptr - delta, *xptr + delta);
	double dS = calc_dS(xptr, &new_xptr);
	// double r = rand_x(local_state, 0, 1);
	// double edS = pow(M_E, -dS);
	if (dS < 0 || pow(M_E, -dS) > rand_x(local_state, 0, 1))
		*xptr = new_xptr;
}

// cuda c kernel
__global__ void Simulate(double *sites, int iterations, hiprandState *state) {
	printf("Kernal launched");
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	for (int _ = 0; _ < iterations; _++) {
		for (int __ = 0; __ < d_n; __++) {
			for (int i = 0; i < d_xjPerThread; i++) {
				int site = idx * d_xjPerThread + i + 1;
				step(sites + site, state + idx);
			}
		}
		__syncthreads();
		//if (idx == 0)
		//printf("S: %lf\n", calc_tot_S(sites));
	}
}

__global__ void print_S(double *sites) {}

__global__ void print_sites(double *sites) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	printf("D: site %i, value %lf\n", idx, sites[idx]);
}

void messure(FILE *file, double *values) {
	for (size_t i = 0; i < MCS; i++) {
		for (size_t k = 0; k < N - 1; k++){
			fprintf(file, "%lf;", values[i * N + k]);
		}
		fprintf(file, "%lf\n", values[i * N + N - 1]);
	}
}

int main() {
	printf("Degrees of freedom N: %ld \n", N);

	printf("Launching.\n"); // setup main configs
	FILE *data_file;
	data_file = fopen(DATA_PATH, "w");
	srand(42);
	// size_t set_limit = size_t(2084) * size_t(2084) * size_t(2084) * 2;
	// printf("test %ld", set_limit);
	// size_t limit = 0;
	// cudaDeviceSetLimit(cudaLimitMallocHeapSize, set_limit);
	// cudaDeviceGetLimit(&limit, cudaLimitMallocHeapSize);
	// printf("Availible memory: %ld\n", limit);
	//  const size_t malloc_limit = size_t()

	// setup cuRAND state for randomization
	printf("Setting cuRAND..\n");
	hiprandState *d_state;
	gpuErrchk(
			hipMalloc((void **)&d_state, threadsPerBlock * blocksPerGrid * sizeof(hiprandState)));
	init_rand<<<blocksPerGrid, threadsPerBlock>>>(d_state);
	gpuErrchk(hipDeviceSynchronize());

	// setup kernel configs
	printf("Threads per block: %ld; Blocks per grid: %ld\n", threadsPerBlock,
			blocksPerGrid);

	// initialise device sites
	double *h_sites, *d_sites;
	gpuErrchk(hipHostMalloc((void **)&h_sites, size, hipHostMallocDefault));
	gpuErrchk(hipMalloc((void **)&d_sites, size));
	//h_sites = (double *)malloc(size);


	clock_t start = clock();
	for (int k = 0; k < runs; k++) {
		// initialise host sites and rand
		for (int i = 0; i < length; i++) {
			*(h_sites + i) = ((double)rand() / RAND_MAX - 0.5) * 20;
		}

		// copy sites from host to device
		gpuErrchk(hipMemcpy(d_sites, h_sites, size, hipMemcpyHostToDevice));

		// wait until the Simulation comes to equilibrium and take first messurement
		Simulate<<<blocksPerGrid, threadsPerBlock>>>(d_sites, RS, d_state);
		//printf("cuda c err: %s\n", cudaGetLastError());
		hipGetLastError();
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(h_sites, d_sites, size, hipMemcpyDeviceToHost));
		messure(data_file, h_sites);

		size_t iterations = MCI / SII - 1;
		for (size_t i = 0; i < iterations; i++) {
			Simulate<<<blocksPerGrid, threadsPerBlock>>>(d_sites, SII, d_state);
			gpuErrchk(hipDeviceSynchronize());
			gpuErrchk(hipMemcpy(h_sites, d_sites, size, hipMemcpyDeviceToHost));
			messure(data_file, h_sites);
			printf("(%3.2lf %%)\n",
					(i + 1 + k * iterations) * 100.0 / iterations / runs);
		}
	}
	clock_t end = clock();
	fclose(data_file);

	// free memory
	printf("Free memory..\n");
	//free(h_sites);
	gpuErrchk(hipHostFree(h_sites));
	gpuErrchk(hipFree(d_sites));

	printf("Done.\n");
	double time_spend = (double)(end - start) / CLOCKS_PER_SEC;
	double benchmark = MCI * MCS * N * runs / time_spend;
	printf("Configurations Messured: %ld; Time Spend on Routine: %0.2lf; "
			"Benchmark: %0.2e sites/s\n",
			total_configurations, time_spend, benchmark);
	return 0;
	;
}
