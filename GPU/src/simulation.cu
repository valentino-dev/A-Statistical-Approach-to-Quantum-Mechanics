#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

// Guide for Device Setting:
// RTX 2060 super has 34 SMs
// 1 Warp are 32 Threads, 64 Warps per SM at maximum
// So 32*64*34=69,632 Threads can be active at once
// 512 or 1024 Threads per Block are ideal, though at 1024 it is harder to start

#define gpuErrchk(ans)                                                         \
{ gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
		bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}

typedef struct{
	int runs, SII, RS; xjPerTHread, n, threadsPerBlock, MCS, MCI, a, m0, lmabda, mu_sq, f_sq, delta, N, total_configurations, blocksPerGrid, array_size;
} setting;


__global__ void init_rand(hiprandState *state) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(42, idx, 0, &state[idx]);
		printf("test\n");
}

__device__ double rand_x(hiprandState *state, double min, double max) {
	return hiprand_uniform(state) * (max - min) + min;
}

__device__ double potential_1(double *x, settings *settings) {
	return settings->mu_sq * pow(*x, 2) * 0.5 + settings->lambda * pow(*x, 4);
}

__device__ double potential_2(double *x, settings *settings) {
	return settings->lambda * pow(*x * *x - settings->f_sq, 2);
}

__device__ double calc_S_of_xj(double x, double *ptr, double *previous_site, double *following_site, setting *settings) {
	return settings->m0 * 0.5 * (pow(*following_site - x, 2) + pow(x - *previous_site, 2)) / settings->a +
		settings->a * (potential_1(&x, settings) + potential_1(ptr - 1, settings));
}

__device__ double calc_dS(double *xptr, double *previous_site, double *following_site, double *newx_ptr, setting *settings) {
	return calc_S_of_xj(*newx_ptr, xptr, previous_site, following_site, settings) - calc_S_of_xj(*xptr, xptr, previous_site, following_site, settings);
}

__device__ void step(double *sites, int site, int previous_site, int following_site, hiprandState *local_state, setting *settings) {
	double *xptr = sites+site;
	double new_x = rand_x(local_state, *xptr - settings->delta, *xptr + settings->delta);
	double dS = calc_dS(xptr, sites+previous_site, sites+following_site, &new_x, settings);
	if (dS < 0 || pow(M_E, -dS) > rand_x(local_state, 0, 1))
		*xptr = new_x;
}

__device__ void Print_Action(double *sites, setting *settings){
	double S = 0;
	for (int i = 0; i < settings->N; i++){	
		S += settings->9 * 0.5 * (pow(sites[(i+1)%N] - sites[i], 2)) / settings->a + settings->a * potential_1(&sites[i], settings);
	}

	printf("S: %lf\n", S);
	
}

// cuda c kernel
__global__ void Simulate(double *sites, int iterations, hiprandState *state, setting *settings) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	for (int _ = 0; _ < iterations; _++) {
		for (int i = 0; i < settings->d_xjPerThread; i++) {
			int site = idx * settings->d_xjPerThread + i;
			int previous_site = (threadIdx.x * settings->d_xjPerThread + i - 1) % (blockDim.x * settings->d_xjPerThread) + blockDim.x * settings->d_xjPerThread * blockIdx.x;
			int following_site = (threadIdx.x * settings->d_xjPerThread + i + 1) % (blockDim.x * settings->d_xjPerThread) + blockDim.x * settings->d_xjPerThread * blockIdx.x;
			for (int __ = 0; __ < settings[4]; __++) {
				step(sites, site, previous_site, following_site, state + idx, settings);
			}
		}
		__syncthreads();
		printf("test\n");
		//if(idx == 0){
			//Print_Action(sites, settings);
				
		//}
	}
}

__global__ void print_sites(double *sites) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	printf("D: site %i, value %lf\n", idx, sites[idx]);
}

void messure(FILE *file, double *values, setting *settings) {
	for (size_t i = 0; i < settings->MCS; i++) {
		for (size_t k = 0; k < setting->N - 1; k++){
			fprintf(file, "%lf;", values[i * N + k]);
		}
		fprintf(file, "%lf\n", values[(i + 1) * N - 1]);
	}
}

__global__ void initial_ensamble(double *sites, hiprandState *state, setting *settings){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	for (int i = 0; i < (int) settings->n; i++) {
		sites[idx * (int) settings->n + i] = rand_x(state + idx, -100, 100);
	}
}

void collectConfigs(config *settings, char **argv){
	settings->runs 			= atof(argv[1]);
	settings->SII 			= atof(argv[2]);
	settings->RS 			= atof(argv[3]);
	settings->xjPerThread 		= atof(argv[4]);
	settings->n 			= atof(argv[5]);
	settings->threadsPerBlock 	= atof(argv[6]);
	settings->MCS 			= atof(argv[7]);
	settings->MCI 			= atof(argv[8]);
	settings->a 			= atof(argv[9]);
	settings->m0 			= atof(argv[10]);
	settings->lambda 		= atof(argv[11]);
	settings->mu_sq 		= atof(argv[12]);
	settings->f_sq 			= atof(argv[13]);

	settings->total_configurations = setting->MCS * setting->runs * setting->MCI / setting->SII;
	settings->array_size = settings->MCS * N * sizeof(double);
	settings->blocksPerGrid = settings->MCS;
	settings->N = settings->xjPerThread * settings->threadsPerBlock;

}


int main(int argc, char** argv) {
	printf("Arguments:");
	for(size_t i = 0; i < argc; i++){
		printf(" %i: %s,", i, argv[i]);
	}
	printf("\n");
	printf("Collecting Settings..\n"); // initialise Settings: data_path, runs, SII, RS, xjPerThread, n, threadsPerBlock, MCS, MCI, a, m0, lambda, mu_sq, f_sq
	config *h_settings, *h_settings;
	gpuErrchk(hipHostMalloc((void **)&h_settings, sizeof(setting)));
	gpuErrchk(hipMalloc((void **)&h_settings, sizeof(setting)));

	collectConfigs(h_settings, argv);

	gpuErrchk(hipMemcpy(d_settings, h_settings, sizeof(setting), hipMemcpyHostToDevice));

	
	//printf("Settings: %i runs, %i SII, %i RS, %i xjPerthread, n=%i, %i threadsPerBlock, %i MCS, %i MCI, a=%lf, m0=%lf, lambda=%lf, mu_sq=%lf, f_sq=%lf\n", runs, SII, RS, xjPerThread, (int) h_settings[4], threadsPerBlock, MCS, MCI, h_settings[8], h_settings[9], h_settings[10], h_settings[11], h_settings[12]);



	printf("Degrees of freedom N: %ld \n", N);

	printf("Launching.\n"); // setup main configs
	FILE *data_file;
	data_file = fopen(argv[1], "w");
	srand(42);


	// setup cuRAND state for randomization
	printf("Setting cuRAND..\n");
	hiprandState *d_state;
	gpuErrchk(
			hipMalloc((void **)&d_state, h_config->threadsPerBlock * h_settings->blocksPerGrid * sizeof(hiprandState)));
	init_rand<<<h_settings->blocksPerGrid, h_conifg->threadsPerBlock>>>(d_state);
	gpuErrchk(hipDeviceSynchronize());

	// setup kernel configs
	printf("Threads per block: %ld; Blocks per grid: %ld\n", h_settings->threadsPerBlock,
			h_settings->blocksPerGrid);

	// initialise device sites
	double *h_sites, *d_sites;
	gpuErrchk(hipHostMalloc((void **)&h_sites, h_settings->array_size));
	gpuErrchk(hipMalloc((void **)&d_sites, h_settings->array_size));
	gpuErrchk(hipMemset(h_sites, 0, h_settings->array_size));
	gpuErrchk(hipMemset(d_sites, 0, h_settings->array_size));


	clock_t start = clock();
	for (int k = 0; k < runs; k++) {
		gpuErrchk(hipMemcpy(d_sites, h_sites, h_settings->array_size, hipMemcpyHostToDevice));
		initial_ensamble<<<h_settings->blocksPerGrid, h_settings->threadsPerBlock>>>(d_sites, d_state, d_settings);

		// wait until the Simulation comes to equilibrium and take first messurement
		Simulate<<<h_settings->blocksPerGrid, h_settings->threadsPerBlock>>>(d_sites, RS, d_state, d_settings);

		size_t iterations = MCI / SII;
		for (size_t i = 0; i < iterations; i++) {
			gpuErrchk(hipDeviceSynchronize());
			gpuErrchk(hipMemcpy(h_sites, d_sites, h_settings->array_size, hipMemcpyDeviceToHost));
			Simulate<<<h_settings->blocksPerGrid, h_settings->threadsPerBlock>>>(d_sites, SII, d_state, d_settings);
			messure(data_file, h_sites, h_settingss);
			printf("(%3.2lf %%)\n",
					(i + 1 + k * iterations) * 100.0 / iterations / runs);
		}
	}
	clock_t end = clock();
	fclose(data_file);
	gpuErrchk(hipDeviceSynchronize());

	// free memory
	printf("Free memory..\n");
	gpuErrchk(hipHostFree(h_sites));
	gpuErrchk(hipFree(d_sites));

	printf("Done.\n");
	double time_spend = (double)(end - start) / CLOCKS_PER_SEC;
	double benchmark = MCI * MCS * N * runs / time_spend;
	printf("Configurations Messured: %ld; Time Spend on Routine: %0.2lfs; "
			"Benchmark: %0.2e sites/s\n",
			total_configurations, time_spend, benchmark);
	return 0;
}
